#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define CUDA_KERNEL_LOOP(1,n) \
for (int i=blockIdx.x*blockDim.x+threadIdx.x; \
i<(n); \
i+=blockDim.x*gridDim.x)

#define FLT_MAX 999999999999

__global__ void relu(const float* A,float* B,const int nthreads)
{
    CUDA_KERNEL_LOOP(index,nthreads){
        if(A[index]>0){
            B[index]=A[index];
        }else{
            B[index]=0;
        }
    }
}